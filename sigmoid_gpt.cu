#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

__global__ void sigmoidActivation(float *z, float *act){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    act[idx] = 1.0f / (1.0f + expf(-z[idx]));
}

int main() {
    int arraySize = 10;
    float h_z[10] = {1,2,3,4,5,6,7,8,9,10};
    float h_act[10];

    float *device_z_values, *device_activations;
    hipMalloc(&device_z_values, arraySize * sizeof(float));
    hipMalloc(&device_activations, arraySize * sizeof(float));

    hipMemcpy(device_z_values, h_z, arraySize * sizeof(float), hipMemcpyHostToDevice);

    // Kernel launch (must be inside main)
    sigmoidActivation<<<1, arraySize>>>(device_z_values, device_activations);

    hipDeviceSynchronize(); // ensure kernel is done

    hipMemcpy(h_act, device_activations, arraySize * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < arraySize; i++) {
        printf("%f -> %f\n", h_z[i], h_act[i]);
    }

    hipFree(device_z_values);
    hipFree(device_activations);

    return 0;
}
